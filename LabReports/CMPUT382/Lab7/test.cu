#include "hip/hip_runtime.h"
__global__ void total(const float *input, float *output, unsigned int len) {
	__shared__ float shared_data[BLOCK_SIZE];

	unsigned int tx = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + tx;

	shared_data[tx] = (i < len) ? input[i] : 0.0f;

	if (i + blockDim.x < len) {
		shared_data[tx] += input[i + blockDim.x];
	}
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tx < s)
		{
			shared_data[tx] += shared_data[tx + s];
		}
		__syncthreads();
	}

	if (tx == 0) {
		output[blockIdx.x] = shared_data[tx];
	}
}
